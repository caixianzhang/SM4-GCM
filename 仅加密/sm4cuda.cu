#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <stdint.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

#include "sm4cuda.cuh"

//S�в���
uint8_t SboxTable[256] = { \
	0xd6,0x90,0xe9,0xfe,0xcc,0xe1,0x3d,0xb7,0x16,0xb6,0x14,0xc2,0x28,0xfb,0x2c,0x05, \
	0x2b,0x67,0x9a,0x76,0x2a,0xbe,0x04,0xc3,0xaa,0x44,0x13,0x26,0x49,0x86,0x06,0x99, \
	0x9c,0x42,0x50,0xf4,0x91,0xef,0x98,0x7a,0x33,0x54,0x0b,0x43,0xed,0xcf,0xac,0x62, \
	0xe4,0xb3,0x1c,0xa9,0xc9,0x08,0xe8,0x95,0x80,0xdf,0x94,0xfa,0x75,0x8f,0x3f,0xa6, \
	0x47,0x07,0xa7,0xfc,0xf3,0x73,0x17,0xba,0x83,0x59,0x3c,0x19,0xe6,0x85,0x4f,0xa8, \
	0x68,0x6b,0x81,0xb2,0x71,0x64,0xda,0x8b,0xf8,0xeb,0x0f,0x4b,0x70,0x56,0x9d,0x35, \
	0x1e,0x24,0x0e,0x5e,0x63,0x58,0xd1,0xa2,0x25,0x22,0x7c,0x3b,0x01,0x21,0x78,0x87, \
	0xd4,0x00,0x46,0x57,0x9f,0xd3,0x27,0x52,0x4c,0x36,0x02,0xe7,0xa0,0xc4,0xc8,0x9e, \
	0xea,0xbf,0x8a,0xd2,0x40,0xc7,0x38,0xb5,0xa3,0xf7,0xf2,0xce,0xf9,0x61,0x15,0xa1, \
	0xe0,0xae,0x5d,0xa4,0x9b,0x34,0x1a,0x55,0xad,0x93,0x32,0x30,0xf5,0x8c,0xb1,0xe3, \
	0x1d,0xf6,0xe2,0x2e,0x82,0x66,0xca,0x60,0xc0,0x29,0x23,0xab,0x0d,0x53,0x4e,0x6f, \
	0xd5,0xdb,0x37,0x45,0xde,0xfd,0x8e,0x2f,0x03,0xff,0x6a,0x72,0x6d,0x6c,0x5b,0x51, \
	0x8d,0x1b,0xaf,0x92,0xbb,0xdd,0xbc,0x7f,0x11,0xd9,0x5c,0x41,0x1f,0x10,0x5a,0xd8, \
	0x0a,0xc1,0x31,0x88,0xa5,0xcd,0x7b,0xbd,0x2d,0x74,0xd0,0x12,0xb8,0xe5,0xb4,0xb0, \
	0x89,0x69,0x97,0x4a,0x0c,0x96,0x77,0x7e,0x65,0xb9,0xf1,0x09,0xc5,0x6e,0xc6,0x84, \
	0x18,0xf0,0x7d,0xec,0x3a,0xdc,0x4d,0x20,0x79,0xee,0x5f,0x3e,0xd7,0xcb,0x39,0x48, \
};

/* System parameter */
uint32_t FK[4] = { 0xa3b1bac6,0x56aa3350,0x677d9197,0xb27022dc };

/* fixed parameter */
uint32_t CK[32] = { \
	0x00070e15,0x1c232a31,0x383f464d,0x545b6269, \
	0x70777e85,0x8c939aa1,0xa8afb6bd,0xc4cbd2d9, \
	0xe0e7eef5,0xfc030a11,0x181f262d,0x343b4249, \
	0x50575e65,0x6c737a81,0x888f969d,0xa4abb2b9, \
	0xc0c7ced5,0xdce3eaf1,0xf8ff060d,0x141b2229, \
	0x30373e45,0x4c535a61,0x686f767d,0x848b9299, \
	0xa0a7aeb5,0xbcc3cad1,0xd8dfe6ed,0xf4fb0209, \
	0x10171e25,0x2c333a41,0x484f565d,0x646b7279, \
};

/*
   ����λ���� C++�汾
   b:��Ҫ�ƶ�������ָ��
   i:��Ҫ�ƶ���λ��
   n:����ֵ��
 */
inline void GET_UINT_BE(uint32_t *n, uint8_t *b, uint32_t i)
{
	(*n) = (((uint32_t)b[i]) << 24) | (((uint32_t)b[i + 1]) << 16) | (((uint32_t)b[i + 2]) << 8) | (uint32_t)b[i + 3];
}

/*
	����λ���� C++�汾������
	b:��Ҫ�ƶ�������ָ��
	i:��Ҫ�ƶ���λ��
	n:����ֵ��
*/
inline void PUT_UINT_BE(uint32_t n, uint8_t *b, uint32_t i)
{
	//ȡn�ĸ���λ
	b[i + 0] = (uint8_t)(n >> 24);

	//ȡn�Ĵθ���λ
	b[i + 1] = (uint8_t)(n >> 16);

	//ȡn�Ĵε���λ
	b[i + 2] = (uint8_t)(n >> 8);

	//ȡn�ĵ���λ
	b[i + 3] = (uint8_t)n;
}

/*
	S���滻
*/
inline uint8_t sm4Sbox(uint8_t inch)
{
	return SboxTable[inch];
}

/*
	ѭ�����ƺ���������xѭ������nλ
*/
inline uint32_t ROTL(uint32_t x, uint32_t n)
{
	return (x << n) | (x >> (32 - n));
}

/*
	����a b��ֵ
*/
inline void SWAP(uint32_t *a, uint32_t *b)
{
	uint32_t c = *a;
	*a = *b;
	*b = c;
}

uint32_t sm4Lt(uint32_t ka)
{
	uint8_t a[4];
	PUT_UINT_BE(ka, a, 0);

	//����滻
	a[0] = sm4Sbox(a[0]);
	a[1] = sm4Sbox(a[1]);
	a[2] = sm4Sbox(a[2]);
	a[3] = sm4Sbox(a[3]);

	//����������ŵ�bb������ȥ
	uint32_t bb = 0;
	GET_UINT_BE(&bb, a, 0);

	//bb�ֱ�����ѭ������2λ��10λ��18λ��24λ������� �õ���ֵ����
	return bb ^ (ROTL(bb, 2)) ^ (ROTL(bb, 10)) ^ (ROTL(bb, 18)) ^ (ROTL(bb, 24));
}

uint32_t sm4F(uint32_t x0, uint32_t x1, uint32_t x2, uint32_t x3, uint32_t rk)
{
	return (x0^sm4Lt(x1^x2^x3^rk));
}


/*
	��Կ��չ����
*/
uint32_t sm4CalciRK(uint32_t ka)
{
	uint8_t a[4];
	PUT_UINT_BE(ka, a, 0);
	a[0] = sm4Sbox(a[0]);
	a[1] = sm4Sbox(a[1]);
	a[2] = sm4Sbox(a[2]);
	a[3] = sm4Sbox(a[3]);

	uint32_t bb = 0;
	GET_UINT_BE(&bb, a, 0);
	return bb ^ (ROTL(bb, 13)) ^ (ROTL(bb, 23));
}

/*
	SK:ֵ���������������д��չ��Կ
	key:��ʼ��Կ(128bit)
*/
void sm4_setkey(uint32_t SK[32], uint8_t key[16])
{
	uint32_t MK[4];
	GET_UINT_BE(&MK[0], key, 0);
	GET_UINT_BE(&MK[1], key, 4);
	GET_UINT_BE(&MK[2], key, 8);
	GET_UINT_BE(&MK[3], key, 12);


	//��ʼ����Կ
	uint32_t k[36];
	k[0] = MK[0] ^ FK[0];
	k[1] = MK[1] ^ FK[1];
	k[2] = MK[2] ^ FK[2];
	k[3] = MK[3] ^ FK[3];

	for (int i = 0; i < 32; i++)
	{
		k[i + 4] = k[i] ^ (sm4CalciRK(k[i + 1] ^ k[i + 2] ^ k[i + 3] ^ CK[i]));
		SK[i] = k[i + 4];
	}
}

/*
	SM4�ֺ���
*/
void sm4_one_round(uint32_t sk[32], uint8_t input[16], uint8_t output[16])
{

	uint32_t ulbuf[36];
	memset(ulbuf, 0, sizeof(ulbuf));

	GET_UINT_BE(&ulbuf[0], input, 0);
	GET_UINT_BE(&ulbuf[1], input, 4);
	GET_UINT_BE(&ulbuf[2], input, 8);
	GET_UINT_BE(&ulbuf[3], input, 12);

	for (int i = 0; i < 32; i++)
	{
		ulbuf[i + 4] = sm4F(ulbuf[i], ulbuf[i + 1], ulbuf[i + 2], ulbuf[i + 3], sk[i]);
	}

	PUT_UINT_BE(ulbuf[35], output, 0);
	PUT_UINT_BE(ulbuf[34], output, 4);
	PUT_UINT_BE(ulbuf[33], output, 8);
	PUT_UINT_BE(ulbuf[32], output, 12);
}

/*
	����ģʽ��Կ��չ
	ctx��ֵ�������������ִ����Ϻ����д������Կ�����Ϣ��
	key: ������Կ������128bit��
*/
void sm4_setkey_enc(sm4_context *ctx, uint8_t key[16])
{
	ctx->mode = SM4_ENCRYPT;
	sm4_setkey(ctx->sk, key);
}

/*
	����ģʽ��Կ��չ
	ctx��ֵ�������������ִ����Ϻ����д������Կ�����Ϣ��
	key: ������Կ������128bit��
*/
void sm4_setkey_dec(sm4_context *ctx, uint8_t key[16])
{
	ctx->mode = SM4_DECRYPT;
	sm4_setkey(ctx->sk, key);
	for (int i = 0; i < 16; i++)
	{
		SWAP(&(ctx->sk[i]), &(ctx->sk[31 - i]));
	}
}

/*
 * SM4-ECB block encryption/decryption
 *
 * SM4-ECBģʽ�ӽ��ܺ���
 * ctx��ֵ�������������Կ����ָ��
 * mode:�ӽ���ģʽ��SM4�����ּӽ���ģʽ�����Ľ������ĳ������Ľ������ĳ�
 * input:��������(16�ֽ�)
 * output:�������(16�ֽ�)
 */
void sm4_crypt_ecb(sm4_context *ctx, int length, uint8_t *input, uint8_t *output)
{
	while (length > 0)
	{
		sm4_one_round(ctx->sk, input, output);
		input += 16;
		output += 16;
		length -= 16;
	}
}

/*
	Ϊ����洢��ô��ͻʱ����Ҫ�����ݿ���д洢ģʽת����
	ת����Ҫ�Ĵ��ڴ�����������󣬹���Ҫ�ĸ����ұ�
	����������ת�����õĲ��ұ�
*/

//ת�����ұ�0
uint32_t matrix_table_zero[32] = {
	 0 * 4 + 0 * 128,  1 * 4 + 0 * 128,  2 * 4 + 0 * 128,  3 * 4 + 0 * 128, \
	 4 * 4 + 0 * 128,  5 * 4 + 0 * 128,  6 * 4 + 0 * 128,  7 * 4 + 0 * 128, \
	 8 * 4 + 1 * 128,  9 * 4 + 1 * 128, 10 * 4 + 1 * 128, 11 * 4 + 1 * 128, \
	12 * 4 + 1 * 128, 13 * 4 + 1 * 128, 14 * 4 + 1 * 128, 15 * 4 + 1 * 128, \
	16 * 4 + 2 * 128, 17 * 4 + 2 * 128, 18 * 4 + 2 * 128, 19 * 4 + 2 * 128, \
	20 * 4 + 2 * 128, 21 * 4 + 2 * 128, 22 * 4 + 2 * 128, 23 * 4 + 2 * 128, \
	24 * 4 + 3 * 128, 25 * 4 + 3 * 128, 26 * 4 + 3 * 128, 27 * 4 + 3 * 128, \
	28 * 4 + 3 * 128, 29 * 4 + 3 * 128, 30 * 4 + 3 * 128, 31 * 4 + 3 * 128, \
};
uint32_t linear_table_zero[32] = {
	 0 * 4 + 0 * 128,  4 * 4 + 0 * 128,  8 * 4 + 0 * 128, 12 * 4 + 0 * 128, \
	 16 * 4 + 0 * 128, 20 * 4 + 0 * 128, 24 * 4 + 0 * 128, 28 * 4 + 0 * 128,\
	 1 * 4 + 1 * 128,  5 * 4 + 1 * 128,  9 * 4 + 1 * 128, 13 * 4 + 1 * 128, \
	17 * 4 + 1 * 128, 21 * 4 + 1 * 128, 25 * 4 + 1 * 128, 29 * 4 + 1 * 128, \
	 2 * 4 + 2 * 128,  6 * 4 + 2 * 128, 10 * 4 + 2 * 128, 14 * 4 + 2 * 128, \
	18 * 4 + 2 * 128, 22 * 4 + 2 * 128, 26 * 4 + 2 * 128, 30 * 4 + 2 * 128, \
	 3 * 4 + 3 * 128,  7 * 4 + 3 * 128, 11 * 4 + 3 * 128, 15 * 4 + 3 * 128, \
	19 * 4 + 3 * 128, 23 * 4 + 3 * 128, 27 * 4 + 3 * 128, 31 * 4 + 3 * 128, \
};
//ת�����ұ�1
uint32_t matrix_table_one[32] = {
	 0 * 4 + 1 * 128,  1 * 4 + 1 * 128,  2 * 4 + 1 * 128,  3 * 4 + 1 * 128, \
	 4 * 4 + 1 * 128,  5 * 4 + 1 * 128,  6 * 4 + 1 * 128,  7 * 4 + 1 * 128, \
	 8 * 4 + 2 * 128,  9 * 4 + 2 * 128, 10 * 4 + 2 * 128, 11 * 4 + 2 * 128, \
	12 * 4 + 2 * 128, 13 * 4 + 2 * 128, 14 * 4 + 2 * 128, 15 * 4 + 2 * 128, \
	16 * 4 + 3 * 128, 17 * 4 + 3 * 128, 18 * 4 + 3 * 128, 19 * 4 + 3 * 128, \
	20 * 4 + 3 * 128, 21 * 4 + 3 * 128, 22 * 4 + 3 * 128, 23 * 4 + 3 * 128, \
	24 * 4 + 0 * 128, 25 * 4 + 0 * 128, 26 * 4 + 0 * 128, 27 * 4 + 0 * 128, \
	28 * 4 + 0 * 128, 29 * 4 + 0 * 128, 30 * 4 + 0 * 128, 31 * 4 + 0 * 128, \
};
uint32_t linear_table_one[32] = {
	 1 * 4 + 0 * 128,  5 * 4 + 0 * 128,  9 * 4 + 0 * 128, 13 * 4 + 0 * 128, \
	 17 * 4 + 0 * 128, 21 * 4 + 0 * 128, 25 * 4 + 0 * 128, 29 * 4 + 0 * 128,\
	 2 * 4 + 1 * 128,  6 * 4 + 1 * 128, 10 * 4 + 1 * 128, 14 * 4 + 1 * 128, \
	18 * 4 + 1 * 128, 22 * 4 + 1 * 128, 26 * 4 + 1 * 128, 30 * 4 + 1 * 128, \
	 3 * 4 + 2 * 128,  7 * 4 + 2 * 128, 11 * 4 + 2 * 128, 15 * 4 + 2 * 128, \
	19 * 4 + 2 * 128, 23 * 4 + 2 * 128, 27 * 4 + 2 * 128, 31 * 4 + 2 * 128, \
	 0 * 4 + 3 * 128,  4 * 4 + 3 * 128,  8 * 4 + 3 * 128, 12 * 4 + 3 * 128, \
	16 * 4 + 3 * 128, 20 * 4 + 3 * 128, 24 * 4 + 3 * 128, 28 * 4 + 3 * 128, \
};
//ת�����ұ�2
uint32_t matrix_table_two[32] = {
	 0 * 4 + 2 * 128,  1 * 4 + 2 * 128,  2 * 4 + 2 * 128,  3 * 4 + 2 * 128, \
	 4 * 4 + 2 * 128,  5 * 4 + 2 * 128,  6 * 4 + 2 * 128,  7 * 4 + 2 * 128, \
	 8 * 4 + 3 * 128,  9 * 4 + 3 * 128, 10 * 4 + 3 * 128, 11 * 4 + 3 * 128, \
	12 * 4 + 3 * 128, 13 * 4 + 3 * 128, 14 * 4 + 3 * 128, 15 * 4 + 3 * 128, \
	16 * 4 + 0 * 128, 17 * 4 + 0 * 128, 18 * 4 + 0 * 128, 19 * 4 + 0 * 128, \
	20 * 4 + 0 * 128, 21 * 4 + 0 * 128, 22 * 4 + 0 * 128, 23 * 4 + 0 * 128, \
	24 * 4 + 1 * 128, 25 * 4 + 1 * 128, 26 * 4 + 1 * 128, 27 * 4 + 1 * 128, \
	28 * 4 + 1 * 128, 29 * 4 + 1 * 128, 30 * 4 + 1 * 128, 31 * 4 + 1 * 128, \
};
uint32_t linear_table_two[32] = {
	 2 * 4 + 0 * 128,  6 * 4 + 0 * 128, 10 * 4 + 0 * 128, 14 * 4 + 0 * 128, \
	 18 * 4 + 0 * 128, 22 * 4 + 0 * 128, 26 * 4 + 0 * 128, 30 * 4 + 0 * 128,\
	 3 * 4 + 1 * 128,  7 * 4 + 1 * 128, 11 * 4 + 1 * 128, 15 * 4 + 1 * 128, \
	19 * 4 + 1 * 128, 23 * 4 + 1 * 128, 27 * 4 + 1 * 128, 31 * 4 + 1 * 128, \
	 0 * 4 + 2 * 128,  4 * 4 + 2 * 128,  8 * 4 + 2 * 128, 12 * 4 + 2 * 128, \
	16 * 4 + 2 * 128, 20 * 4 + 2 * 128, 24 * 4 + 2 * 128, 28 * 4 + 2 * 128, \
	 1 * 4 + 3 * 128,  5 * 4 + 3 * 128,  9 * 4 + 3 * 128, 13 * 4 + 3 * 128, \
	17 * 4 + 3 * 128, 21 * 4 + 3 * 128, 25 * 4 + 3 * 128, 29 * 4 + 3 * 128, \
};
//ת�����ұ�3
uint32_t matrix_table_three[32] = {
	 0 * 4 + 3 * 128,  1 * 4 + 3 * 128,  2 * 4 + 3 * 128,  3 * 4 + 3 * 128, \
	 4 * 4 + 3 * 128,  5 * 4 + 3 * 128,  6 * 4 + 3 * 128,  7 * 4 + 3 * 128, \
	 8 * 4 + 0 * 128,  9 * 4 + 0 * 128, 10 * 4 + 0 * 128, 11 * 4 + 0 * 128, \
	12 * 4 + 0 * 128, 13 * 4 + 0 * 128, 14 * 4 + 0 * 128, 15 * 4 + 0 * 128, \
	16 * 4 + 1 * 128, 17 * 4 + 1 * 128, 18 * 4 + 1 * 128, 19 * 4 + 1 * 128, \
	20 * 4 + 1 * 128, 21 * 4 + 1 * 128, 22 * 4 + 1 * 128, 23 * 4 + 1 * 128, \
	24 * 4 + 2 * 128, 25 * 4 + 2 * 128, 26 * 4 + 2 * 128, 27 * 4 + 2 * 128, \
	28 * 4 + 2 * 128, 29 * 4 + 2 * 128, 30 * 4 + 2 * 128, 31 * 4 + 2 * 128, \
};
uint32_t linear_table_three[32] = {
	 3 * 4 + 0 * 128,  7 * 4 + 0 * 128, 11 * 4 + 0 * 128, 15 * 4 + 0 * 128, \
	 19 * 4 + 0 * 128, 23 * 4 + 0 * 128, 27 * 4 + 0 * 128, 31 * 4 + 0 * 128,\
	 0 * 4 + 1 * 128,  4 * 4 + 1 * 128,  8 * 4 + 1 * 128, 12 * 4 + 1 * 128, \
	16 * 4 + 1 * 128, 20 * 4 + 1 * 128, 24 * 4 + 1 * 128, 28 * 4 + 1 * 128, \
	 1 * 4 + 2 * 128,  5 * 4 + 2 * 128,  9 * 4 + 2 * 128, 13 * 4 + 2 * 128, \
	17 * 4 + 2 * 128, 21 * 4 + 2 * 128, 25 * 4 + 2 * 128, 29 * 4 + 2 * 128, \
	 2 * 4 + 3 * 128,  6 * 4 + 3 * 128, 10 * 4 + 3 * 128, 14 * 4 + 3 * 128, \
	18 * 4 + 3 * 128, 22 * 4 + 3 * 128, 26 * 4 + 3 * 128, 30 * 4 + 3 * 128, \
};

//ÿ���߳̿鹲��IV, SK, ency0, lenAC
__constant__ uint8_t constant_iv[12];
__constant__ uint32_t constant_sk[32];
__constant__ uint8_t  constant_ency0[16];
__constant__ uint8_t  constant_lenAC[16];

void otherT(uint8_t T[16][256][16])
{
	int i = 0, j = 0, k = 0;
	uint64_t vh, vl;
	uint64_t zh, zl;
	for (i = 0; i < 256; i++)
	{
		vh = ((uint64_t)T[0][i][0] << 56) ^ ((uint64_t)T[0][i][1] << 48) ^ \
			((uint64_t)T[0][i][2] << 40) ^ ((uint64_t)T[0][i][3] << 32) ^ \
			((uint64_t)T[0][i][4] << 24) ^ ((uint64_t)T[0][i][5] << 16) ^ \
			((uint64_t)T[0][i][6] << 8) ^ ((uint64_t)T[0][i][7]);

		vl = ((uint64_t)T[0][i][8] << 56) ^ ((uint64_t)T[0][i][9] << 48) ^ \
			((uint64_t)T[0][i][10] << 40) ^ ((uint64_t)T[0][i][11] << 32) ^ \
			((uint64_t)T[0][i][12] << 24) ^ ((uint64_t)T[0][i][13] << 16) ^ \
			((uint64_t)T[0][i][14] << 8) ^ ((uint64_t)T[0][i][15]);

		zh = zl = 0;

		for (j = 0; j <= 120; j++)
		{
			if ((j > 0) && (0 == j % 8))
			{
				zh ^= vh;
				zl ^= vl;
				for (k = 1; k <= 16 / 2; k++)
				{
					T[j / 8][i][16 / 2 - k] = (uint8_t)zh;
					zh = zh >> 8;
					T[j / 8][i][16 - k] = (uint8_t)zl;
					zl = zl >> 8;
				}
				zh = zl = 0;
			}
			if (vl & 0x1)
			{
				vl = vl >> 1;
				if (vh & 0x1) { vl ^= 0x8000000000000000; }
				vh = vh >> 1;
				vh ^= 0xe100000000000000;
			}
			else
			{
				vl = vl >> 1;
				if (vh & 0x1) { vl ^= 0x8000000000000000; }
				vh = vh >> 1;
			}
		}
	}
}

//����GF�˷���
void computeTable(uint8_t T[16][256][16], uint8_t H[16])
{
	// zh is the higher 64-bit, zl is the lower 64-bit
	uint64_t zh = 0, zl = 0;
	// vh is the higher 64-bit, vl is the lower 64-bit
	uint64_t vh = ((uint64_t)H[0] << 56) ^ ((uint64_t)H[1] << 48) ^ \
		((uint64_t)H[2] << 40) ^ ((uint64_t)H[3] << 32) ^ \
		((uint64_t)H[4] << 24) ^ ((uint64_t)H[5] << 16) ^ \
		((uint64_t)H[6] << 8) ^ ((uint64_t)H[7]);

	uint64_t vl = ((uint64_t)H[8] << 56) ^ ((uint64_t)H[9] << 48) ^ \
		((uint64_t)H[10] << 40) ^ ((uint64_t)H[11] << 32) ^ \
		((uint64_t)H[12] << 24) ^ ((uint64_t)H[13] << 16) ^ \
		((uint64_t)H[14] << 8) ^ ((uint64_t)H[15]);

	uint8_t temph;

	uint64_t tempvh = vh;
	uint64_t tempvl = vl;
	int i = 0, j = 0;
	for (i = 0; i < 256; i++)
	{
		temph = (uint8_t)i;
		vh = tempvh;
		vl = tempvl;
		zh = zl = 0;

		for (j = 0; j < 8; j++)
		{
			if (0x80 & temph)
			{
				zh ^= vh;
				zl ^= vl;
			}
			if (vl & 0x1)
			{
				vl = vl >> 1;
				if (vh & 0x1) { vl ^= 0x8000000000000000; }
				vh = vh >> 1;
				vh ^= 0xe100000000000000;
			}
			else
			{
				vl = vl >> 1;
				if (vh & 0x1) { vl ^= 0x8000000000000000; }
				vh = vh >> 1;
			}
			temph = temph << 1;
		}
		// get result
		for (j = 1; j <= 16 / 2; j++)
		{
			T[0][i][16 / 2 - j] = (uint8_t)zh;
			zh = zh >> 8;
			T[0][i][16 - j] = (uint8_t)zl;
			zl = zl >> 8;
		}
	}
	otherT(T);
}

/**
 * return the value of (output.H) by looking up tables
 */
void multi(uint8_t T[16][256][16], uint8_t *output)
{
	uint8_t i, j;
	uint8_t temp[16];
	for (i = 0; i < 16; i++)
	{
		temp[i] = output[i];
		output[i] = 0;
	}
	for (i = 0; i < 16; i++)
	{
		for (j = 0; j < 16; j++)
		{
			output[j] ^= T[i][*(temp + i)][j];
		}
	}
}

/*
 * a: additional authenticated data
 * c: the cipher text or initial vector
 */
void ghash(uint8_t T[16][256][16], uint8_t *add, size_t add_len, uint8_t *cipher, size_t length, uint8_t *output)
{
	/* x0 = 0 */
	*(uint64_t *)output = 0;
	*((uint64_t *)output + 1) = 0;

	/* compute with add */
	int i = 0;
	for (i = 0; i < add_len / 16; i++)
	{
		*(uint64_t *)output ^= *(uint64_t *)add;
		*((uint64_t *)output + 1) ^= *((uint64_t *)add + 1);
		add += 16;
		multi(T, output);
	}

	if (add_len % 16)
	{
		// the remaining add
		for (i = 0; i < add_len % 16; i++)
		{
			*(output + i) ^= *(add + i);
		}
		multi(T, output);
	}

	/* compute with cipher text */
	for (i = 0; i < length / 16; i++)
	{
		*(uint64_t *)output ^= *(uint64_t *)cipher;
		*((uint64_t *)output + 1) ^= *((uint64_t *)cipher + 1);
		cipher += 16;
		multi(T, output);
	}
	if (length % 16)
	{
		// the remaining cipher
		for (i = 0; i < length % 16; i++)
		{
			*(output + i) ^= *(cipher + i);
		}
		multi(T, output);
	}

	/* eor (len(A)||len(C)) */
	uint64_t temp_len = (uint64_t)(add_len * 8); // len(A) = (uint64_t)(add_len*8)
	for (i = 1; i <= 16 / 2; i++)
	{
		output[16 / 2 - i] ^= (uint8_t)temp_len;
		temp_len = temp_len >> 8;
	}
	temp_len = (uint64_t)(length * 8); // len(C) = (uint64_t)(length*8)
	for (i = 1; i <= 16 / 2; i++)
	{
		output[16 - i] ^= (uint8_t)temp_len;
		temp_len = temp_len >> 8;
	}
	multi(T, output);
}

/*
**	���˺��������������Դ洢ģʽת��Ϊ���δ洢ģʽ
**	dev_linear�����Դ洢ģʽ���ݿ飬�����ݿ���������ʽ�洢��ȫ���ڴ���
**	dev_matrix�����δ洢ģʽ���ݿ飬�����ݿ��Ծ�����ʽ�洢��ȫ���ڴ���
*/
__global__ void kernal_linear_to_matrix(\
	uint32_t dev_matrix_table_zero[32], uint32_t dev_linear_table_zero[32], \
	uint32_t dev_matrix_table_one[32], uint32_t dev_linear_table_one[32], \
	uint32_t dev_matrix_table_two[32], uint32_t dev_linear_table_two[32], \
	uint32_t dev_matrix_table_three[32], uint32_t dev_linear_table_three[32], \
	uint8_t dev_linear[PARTICLE_SIZE / STREAM_SIZE], \
	uint8_t dev_matrix[PARTICLE_SIZE / STREAM_SIZE])
{
	__shared__ uint8_t smem[16 * BLOCK_SIZE * 2];
	uint8_t *matrix = smem;
	uint8_t *linear = smem + 16 * BLOCK_SIZE;

	uint32_t dev_offset = blockIdx.x * blockDim.x * 16 + threadIdx.x * 4;
	uint32_t share_offset = threadIdx.x * 4;

	//�Զ���ϲ��ô�ķ�ʽ�����ݴ�ȫ���ڴ滺�浽�����ڴ�
	{
		uint32_t *read = (uint32_t *)(dev_linear + dev_offset);
		uint32_t *write = (uint32_t *)(linear + share_offset);

		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(write + i * BLOCK_SIZE) = *(read + i * BLOCK_SIZE);
		}
	}

	//ͬ����
	__syncthreads();

	//���ת��
	{
		uint32_t warpaddr = (threadIdx.x / 32) * (32 * 16);
		uint32_t inertid = threadIdx.x % 32;
		uint32_t *read;
		uint32_t *write;

		//��0��ת��
		write = (uint32_t *)(matrix + warpaddr + dev_matrix_table_zero[inertid]);
		read = (uint32_t *)(linear + warpaddr + dev_linear_table_zero[inertid]);
		*write = *read;

		//��1��ת��
		write = (uint32_t *)(matrix + warpaddr + dev_matrix_table_one[inertid]);
		read = (uint32_t *)(linear + warpaddr + dev_linear_table_one[inertid]);
		*write = *read;

		//��2��ת��
		write = (uint32_t *)(matrix + warpaddr + dev_matrix_table_two[inertid]);
		read = (uint32_t *)(linear + warpaddr + dev_linear_table_two[inertid]);
		*write = *read;

		//��3��ת��
		write = (uint32_t *)(matrix + warpaddr + dev_matrix_table_three[inertid]);
		read = (uint32_t *)(linear + warpaddr + dev_linear_table_three[inertid]);
		*write = *read;
	}

	//ͬ����
	__syncthreads();

	//�Զ���ϲ��ô�ķ�ʽ�����ݴӹ����ڴ�д��ȫ���ڴ�
	{
		uint32_t *write = (uint32_t *)(dev_matrix + dev_offset);
		uint32_t *read = (uint32_t *)(matrix + share_offset);

		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(write + i * BLOCK_SIZE) = *(read + i * BLOCK_SIZE);
		}
	}
}

/*
**	���˺����������ɾ��δ洢ģʽת��Ϊ���Դ洢ģʽ
**	dev_matrix�����δ洢ģʽ���ݿ飬�����ݿ��Ծ�����ʽ�洢��ȫ���ڴ���
**	dev_linear�����Դ洢ģʽ���ݿ飬�����ݿ���������ʽ�洢��ȫ���ڴ���
*/
__global__ void kernal_matrix_to_linear(\
	uint32_t dev_matrix_table_zero[32], uint32_t dev_linear_table_zero[32], \
	uint32_t dev_matrix_table_one[32], uint32_t dev_linear_table_one[32], \
	uint32_t dev_matrix_table_two[32], uint32_t dev_linear_table_two[32], \
	uint32_t dev_matrix_table_three[32], uint32_t dev_linear_table_three[32], \
	uint8_t dev_matrix[PARTICLE_SIZE / STREAM_SIZE], \
	uint8_t dev_linear[PARTICLE_SIZE / STREAM_SIZE])
{
	__shared__ uint8_t smem[16 * BLOCK_SIZE * 2];
	uint8_t *matrix = smem;
	uint8_t *linear = smem + 16 * BLOCK_SIZE;
	uint32_t dev_offset = blockIdx.x * blockDim.x * 16 + threadIdx.x * 4;
	uint32_t share_offset = threadIdx.x * 4;

	//�Զ���ϲ��ô�ķ�ʽ�����ݴ�ȫ���ڴ滺�浽�����ڴ�
	{
		uint32_t *read = (uint32_t *)(dev_matrix + dev_offset);
		uint32_t *write = (uint32_t *)(matrix + share_offset);

		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(write + i * BLOCK_SIZE) = *(read + i * BLOCK_SIZE);
		}
	}

	//ͬ����
	__syncthreads();

	//���ת��
	{
		uint32_t warpaddr = (threadIdx.x / 32) * (32 * 16);
		uint32_t inertid = threadIdx.x % 32;
		uint32_t *read;
		uint32_t *write;

		//��0��ת��
		read = (uint32_t *)(matrix + warpaddr + dev_matrix_table_zero[inertid]);
		write = (uint32_t *)(linear + warpaddr + dev_linear_table_zero[inertid]);
		*write = *read;

		//��1��ת��
		read = (uint32_t *)(matrix + warpaddr + dev_matrix_table_one[inertid]);
		write = (uint32_t *)(linear + warpaddr + dev_linear_table_one[inertid]);
		*write = *read;

		//��2��ת��
		read = (uint32_t *)(matrix + warpaddr + dev_matrix_table_two[inertid]);
		write = (uint32_t *)(linear + warpaddr + dev_linear_table_two[inertid]);
		*write = *read;

		//��3��ת��
		read = (uint32_t *)(matrix + warpaddr + dev_matrix_table_three[inertid]);
		write = (uint32_t *)(linear + warpaddr + dev_linear_table_three[inertid]);
		*write = *read;
	}

	//ͬ����
	__syncthreads();

	//�Զ���ϲ��ô�ķ�ʽ�����ݴӹ����ڴ�д��ȫ���ڴ�
	{
		uint32_t *write = (uint32_t *)(dev_linear + dev_offset);
		uint32_t *read = (uint32_t *)(linear + share_offset);

		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(write + i * BLOCK_SIZE) = *(read + i * BLOCK_SIZE);
		}
	}
}

/*
**	�����㷨�˺��������SM4-CTRģʽ���ܣ�ÿ���̼߳���һ����ţ�֮�����������ݿ������������
**	dev_SboxTable:S��
**	counter:���ݿ����
**	streamid:��ID
**	dev_input:������������
**	dev_output:�����������
*/
__global__ void kernal_enc(uint8_t *const __restrict__ dev_SboxTable, \
	uint32_t dev_matrix_table_zero[32], uint32_t dev_linear_table_zero[32], \
	uint32_t dev_matrix_table_one[32], uint32_t dev_linear_table_one[32], \
	uint32_t dev_matrix_table_two[32], uint32_t dev_linear_table_two[32], \
	uint32_t dev_matrix_table_three[32], uint32_t dev_linear_table_three[32], \
	uint32_t counter, uint32_t streamid, \
	uint8_t dev_input[PARTICLE_SIZE / STREAM_SIZE], \
	uint8_t dev_output[PARTICLE_SIZE / STREAM_SIZE])
{
	__shared__ uint8_t smem[16 * BLOCK_SIZE * 2];
	uint8_t *matrix = smem;
	uint8_t *linear = smem + 16 * BLOCK_SIZE;
	uint8_t *rw_matrix = matrix + (threadIdx.x / 32) * (16 * 32) + (threadIdx.x % 32) * 4;
	uint32_t dev_offset = blockIdx.x * blockDim.x * 16 + threadIdx.x * 4;
	uint32_t share_offset = threadIdx.x * 4;

	{
		uint32_t ulbuf[5];

		{
			//�����̶߳�ȡiv
			uint8_t tidCTR[16];

			*(uint32_t *)(tidCTR + 0) = *(uint32_t *)(constant_iv + 0);
			*(uint32_t *)(tidCTR + 4) = *(uint32_t *)(constant_iv + 4);
			*(uint32_t *)(tidCTR + 8) = *(uint32_t *)(constant_iv + 8);

			*(uint32_t *)(tidCTR + 12) = counter + (uint32_t)(threadIdx.x + blockIdx.x * blockDim.x + streamid * (PARTICLE_SIZE / STREAM_SIZE / 16));
			//*(uint32_t *)(tidCTR + 12) = counter;

			#pragma unroll 4
			for (int i = 0; i < 4; i++)
			{
				ulbuf[i] = (((uint32_t)tidCTR[i * 4]) << 24) | \
					(((uint32_t)tidCTR[i * 4 + 1]) << 16) | \
					(((uint32_t)tidCTR[i * 4 + 2]) << 8) | \
					(uint32_t)tidCTR[i * 4 + 3];
			}
		}

		//32�ֵ�������
		{
			uint32_t temp;
			uint8_t a[4];
			uint32_t bb;

			#pragma unroll 32
			for (int i = 0; i < 32; i++)
			{
				temp = ulbuf[(i + 1) % 5] ^ ulbuf[(i + 2) % 5] ^ ulbuf[(i + 3) % 5] ^ constant_sk[i];
				a[0] = (uint8_t)(temp >> 24);
				a[1] = (uint8_t)(temp >> 16);
				a[2] = (uint8_t)(temp >> 8);
				a[3] = (uint8_t)temp;
				a[0] = dev_SboxTable[a[0]];
				a[1] = dev_SboxTable[a[1]];
				a[2] = dev_SboxTable[a[2]];
				a[3] = dev_SboxTable[a[3]];
				bb = (((uint32_t)a[0]) << 24) | (((uint32_t)a[1]) << 16) | (((uint32_t)a[2]) << 8) | (uint32_t)a[3];
				bb = bb ^ ((bb << 2) | (bb >> 30)) ^ ((bb << 10) | (bb >> 22)) ^ ((bb << 18) | (bb >> 14)) ^ ((bb << 24) | (bb >> 8));
				ulbuf[(i + 4) % 5] = ulbuf[(i + 0) % 5] ^ bb;
			}
		}

		{
			//��д���߳����������ʼ��ַ(���δ洢ģʽ)�����Ĵ���ڹ����ڴ�
			uint8_t temp[4];
			uint8_t *write = rw_matrix;

			temp[0] = (uint8_t)(ulbuf[0] >> 24);
			temp[1] = (uint8_t)(ulbuf[0] >> 16);
			temp[2] = (uint8_t)(ulbuf[0] >> 8);
			temp[3] = (uint8_t)ulbuf[0];
			*(uint32_t *)(rw_matrix + 0 * 128) = *(uint32_t *)temp;

			temp[0] = (uint8_t)(ulbuf[4] >> 24);
			temp[1] = (uint8_t)(ulbuf[4] >> 16);
			temp[2] = (uint8_t)(ulbuf[4] >> 8);
			temp[3] = (uint8_t)ulbuf[4];
			*(uint32_t *)(rw_matrix + 1 * 128) = *(uint32_t *)temp;

			temp[0] = (uint8_t)(ulbuf[3] >> 24);
			temp[1] = (uint8_t)(ulbuf[3] >> 16);
			temp[2] = (uint8_t)(ulbuf[3] >> 8);
			temp[3] = (uint8_t)ulbuf[3];
			*(uint32_t *)(rw_matrix + 2 * 128) = *(uint32_t *)temp;

			temp[0] = (uint8_t)(ulbuf[2] >> 24);
			temp[1] = (uint8_t)(ulbuf[2] >> 16);
			temp[2] = (uint8_t)(ulbuf[2] >> 8);
			temp[3] = (uint8_t)ulbuf[2];
			*(uint32_t *)(rw_matrix + 3 * 128) = *(uint32_t *)temp;
		}
	}

	//ͬ����
	__syncthreads();

	//�������ڴ��о��δ洢ģʽ������ת�������Դ洢ģʽ
	{
		uint32_t warpaddr = (threadIdx.x / 32) * (32 * 16);
		uint32_t inertid = threadIdx.x % 32;
		uint32_t *read;
		uint32_t *write;

		//��0��ת��
		read = (uint32_t *)(matrix + warpaddr + dev_matrix_table_zero[inertid]);
		write = (uint32_t *)(linear + warpaddr + dev_linear_table_zero[inertid]);
		*write = *read;

		//��1��ת��
		read = (uint32_t *)(matrix + warpaddr + dev_matrix_table_one[inertid]);
		write = (uint32_t *)(linear + warpaddr + dev_linear_table_one[inertid]);
		*write = *read;

		//��2��ת��
		read = (uint32_t *)(matrix + warpaddr + dev_matrix_table_two[inertid]);
		write = (uint32_t *)(linear + warpaddr + dev_linear_table_two[inertid]);
		*write = *read;

		//��3��ת��
		read = (uint32_t *)(matrix + warpaddr + dev_matrix_table_three[inertid]);
		write = (uint32_t *)(linear + warpaddr + dev_linear_table_three[inertid]);
		*write = *read;
	}

	//ͬ����
	__syncthreads();

	//�Զ���ϲ��ô�ķ�ʽ��ȡ���ģ������ܺ����������������������ģ��������ĺ����Զ���ϲ��ô�ķ�ʽд��ȫ���ڴ�
	{
		uint32_t *read = (uint32_t *)(dev_input + dev_offset);
		uint32_t *write = (uint32_t *)(dev_output + dev_offset);
		uint32_t *cipher = (uint32_t *)(linear + share_offset);

		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(write + i * BLOCK_SIZE) = (*(read + i * BLOCK_SIZE)) ^ (*(cipher + i * BLOCK_SIZE));
		}
	}
}
/*
**	������˷��ӷ�����˺���
**	dev_gfmult_table:������˷���
**	dev_cipher:��������(���δ洢ģʽ)
**	dev_gfmult:������˷����(���δ洢ģʽ)
*/
__global__ void kernal_gfmult(\
	uint8_t dev_gfmult_table[16][256][16], \
	uint8_t dev_cipher[PARTICLE_SIZE / STREAM_SIZE], \
	uint8_t dev_gfmult[PARTICLE_SIZE / STREAM_SIZE])
{
	__shared__ uint8_t smem[16 * BLOCK_SIZE];
	uint8_t *matrix = smem;

	uint32_t dev_offset = blockIdx.x * blockDim.x * 16 + threadIdx.x * 4;
	uint32_t share_offset = threadIdx.x * 4;

	//�Զ���ϲ��ô�ķ�ʽ��ȫ���ڴ��ȡ��������һ��������˷����������������Ľ��д�������ڴ�
	//��ʱ�����ڴ��е����ݿ��Ծ��δ洢ģʽ�洢��
	{
		uint32_t *read_cipher = (uint32_t *)(dev_cipher + dev_offset);
		uint32_t *read_gfmult = (uint32_t *)(dev_gfmult + dev_offset);
		uint32_t *write = (uint32_t *)(matrix + share_offset);

		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(write + i * BLOCK_SIZE) = (*(read_cipher + i * BLOCK_SIZE)) ^ (*(read_gfmult + i * BLOCK_SIZE));
		}
	}

	//ͬ����
	__syncthreads();

	//������˷�
	{
		uint8_t *tid_cipher = matrix + (threadIdx.x / 32) * (16 * 32) + (threadIdx.x % 32) * 4;
		uint8_t temp;
		uint8_t *read;

		//�ݴ�GF�˷����
		uint8_t tid_gfmult[16];
		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(uint32_t *)(tid_gfmult + i * 4) = 0;
		}

		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			read = tid_cipher + i * (32 * 4);

			#pragma unroll 4
			for (int j = 0; j < 4; j++)
			{
				temp = read[j];

				#pragma unroll 16
				for (int k = 0; k < 16; k++)
				{
					tid_gfmult[k] ^= dev_gfmult_table[i * 4 + j][temp][k];
				}
			}
		}

		//�������ݿ��������˷��Ľ��д�ع����ڴ�
		{
			uint32_t *write = (uint32_t *)(matrix + (threadIdx.x / 32) * (16 * 32) + (threadIdx.x % 32) * 4);

			#pragma unroll 4
			for (int i = 0; i < 4; i++)
			{
				*(write + i * 32) = *(uint32_t *)(tid_gfmult + i * 4);
			}
		}
	}

	//ͬ����
	__syncthreads();

	//�Զ���ϲ��ô�ķ�ʽ�������ڴ��еĳ˷����д��ȫ���ڴ棬��ʱ���ݿ��Ծ��δ洢ģʽ�����ȫ���ڴ��С�
	{
		uint32_t *write = (uint32_t *)(dev_gfmult + dev_offset);
		uint32_t *read = (uint32_t *)(matrix + share_offset);

		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(write + i * BLOCK_SIZE) = *(read + i * BLOCK_SIZE);
		}
	}
}

/*
**	���˺�����ɼ���ÿ���߳����յ�GHASH���
**	dev_gfmult_table;������˷���
**	dev_gfmult:������˷����
*/
__global__ void kernal_final(\
	uint8_t dev_gfmult_table[16][256][16], \
	uint32_t dev_matrix_table_zero[32], uint32_t dev_linear_table_zero[32], \
	uint32_t dev_matrix_table_one[32], uint32_t dev_linear_table_one[32], \
	uint32_t dev_matrix_table_two[32], uint32_t dev_linear_table_two[32], \
	uint32_t dev_matrix_table_three[32], uint32_t dev_linear_table_three[32], \
	uint8_t dev_gfmult[PARTICLE_SIZE / STREAM_SIZE])
{
	__shared__ uint8_t smem[16 * BLOCK_SIZE];
	uint8_t *matrix = smem;
	uint32_t dev_offset = blockIdx.x * blockDim.x * 16 + threadIdx.x * 4;
	uint32_t share_offset = threadIdx.x * 4;
	//�Զ���ϲ��ô淽ʽ��ȡǰһ��GF�˷�����������ڴ�
	{
		uint32_t *read = (uint32_t *)(dev_gfmult + dev_offset);
		uint32_t *write = (uint32_t *)(matrix + share_offset);

		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(write + i * BLOCK_SIZE) = *(read + i * BLOCK_SIZE);
		}
	}

	//ͬ����
	__syncthreads();

	{
		uint8_t *tid_cipher = matrix + (threadIdx.x / 32) * (16 * 32) + (threadIdx.x % 32) * 4;
		uint8_t temp;
		uint8_t *read;

		//�ݴ�GF�˷��м���
		uint8_t tid_gfmult[16];
		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(uint32_t *)(tid_gfmult + i * 4) = 0;
		}

		//�������������˷�
		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			read = tid_cipher + i * (32 * 4);

			#pragma unroll 4
			for (int j = 0; j < 4; j++)
			{
				temp = read[j] ^ constant_lenAC[i * 4 + j];

				#pragma unroll 16
				for (int k = 0; k < 16; k++)
				{
					tid_gfmult[k] ^= dev_gfmult_table[i * 4 + j][temp][k];
				}
			}
		}

		//ÿ���߳���ency0����������յ�tag
		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(uint32_t *)(tid_gfmult + i * 4) ^= *(uint32_t *)(constant_ency0 + i * 4);
		}

		//�Զ���ϲ��ô�ķ�ʽ������GF�˷����д�ع����ڴ棬��ʱ���ݿ��Ծ��δ洢ģʽ����ڹ����ڴ���
		{
			uint32_t *write = (uint32_t *)(matrix + (threadIdx.x / 32) * (16 * 32) + (threadIdx.x % 32) * 4);

			#pragma unroll 4
			for (int i = 0; i < 4; i++)
			{
				*(write + i * 32) = *(uint32_t *)(tid_gfmult + i * 4);
			}
		}
	}

	//ͬ����
	__syncthreads();

	//�������ڴ��еĳ˷��������ϲ��ô�ķ�ʽд��ȫ���ڴ�
	{
		uint32_t *write = (uint32_t *)(dev_gfmult + dev_offset);
		uint32_t *read = (uint32_t *)(matrix + share_offset);

		#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			*(write + i * BLOCK_SIZE) = *(read + i * BLOCK_SIZE);
		}
	}
}


void Init_device_memory(device_memory *way, uint8_t add[16], uint8_t iv[12])
{
	//������
	for (int i = 0; i < STREAM_SIZE; i++)
	{
		hipStreamCreate(&(way->stream[i]));
	}

	//��ʼ���洢ģʽת�����ұ��ڴ�ռ�
	hipHostAlloc((void**)&(way->dev_matrix_table_zero), 32 * sizeof(uint32_t), hipHostMallocDefault);
	hipMemcpy(way->dev_matrix_table_zero, matrix_table_zero, 32 * sizeof(uint32_t), hipMemcpyHostToDevice);

	hipHostAlloc((void**)&(way->dev_linear_table_zero), 32 * sizeof(uint32_t), hipHostMallocDefault);
	hipMemcpy(way->dev_linear_table_zero, linear_table_zero, 32 * sizeof(uint32_t), hipMemcpyHostToDevice);

	hipHostAlloc((void**)&(way->dev_matrix_table_one), 32 * sizeof(uint32_t), hipHostMallocDefault);
	hipMemcpy(way->dev_matrix_table_one, matrix_table_one, 32 * sizeof(uint32_t), hipMemcpyHostToDevice);

	hipHostAlloc((void**)&(way->dev_linear_table_one), 32 * sizeof(uint32_t), hipHostMallocDefault);
	hipMemcpy(way->dev_linear_table_one, linear_table_one, 32 * sizeof(uint32_t), hipMemcpyHostToDevice);

	hipHostAlloc((void**)&(way->dev_matrix_table_two), 32 * sizeof(uint32_t), hipHostMallocDefault);
	hipMemcpy(way->dev_matrix_table_two, matrix_table_two, 32 * sizeof(uint32_t), hipMemcpyHostToDevice);

	hipHostAlloc((void**)&(way->dev_linear_table_two), 32 * sizeof(uint32_t), hipHostMallocDefault);
	hipMemcpy(way->dev_linear_table_two, linear_table_two, 32 * sizeof(uint32_t), hipMemcpyHostToDevice);

	hipHostAlloc((void**)&(way->dev_matrix_table_three), 32 * sizeof(uint32_t), hipHostMallocDefault);
	hipMemcpy(way->dev_matrix_table_three, matrix_table_three, 32 * sizeof(uint32_t), hipMemcpyHostToDevice);

	hipHostAlloc((void**)&(way->dev_linear_table_three), 32 * sizeof(uint32_t), hipHostMallocDefault);
	hipMemcpy(way->dev_linear_table_three, linear_table_three, 32 * sizeof(uint32_t), hipMemcpyHostToDevice);

	//������Կ�����������ڴ�
	hipMemcpyToSymbol(HIP_SYMBOL(constant_sk), way->ctx.sk, 32 * sizeof(uint32_t));

	//��ʼ��ÿ���̵߳�IV
	hipMemcpyToSymbol(HIP_SYMBOL(constant_iv), iv, 12);
	
	//��ʼ��S���ڴ�ռ�
	hipHostAlloc((void**)&(way->dev_SboxTable), 256, hipHostMallocDefault);
	hipMemcpy(way->dev_SboxTable, SboxTable, 256, hipMemcpyHostToDevice);

	//������������ռ�
	hipHostAlloc((void**)&(way->dev_input), PARTICLE_SIZE, hipHostMallocDefault);

	//������������ռ�
	hipHostAlloc((void**)&(way->dev_output), PARTICLE_SIZE, hipHostMallocDefault);

	//����ȫ0���Ŀ�
	uint8_t y0[16];
	uint8_t ency0[16];
	memset(y0, 0, 16);

	//��ency0�����������ڴ�
	sm4_crypt_ecb(&way->ctx, 16, y0, ency0);
	hipMemcpyToSymbol(HIP_SYMBOL(constant_ency0), ency0, 16);

	uint8_t gfmult_table[16][256][16];
	//����������˷����ұ�
	computeTable(gfmult_table, ency0);

	//��������˷�������ȫ���ڴ�
	hipHostAlloc((void**)&(way->dev_gfmult_table), \
		sizeof(gfmult_table), hipHostMallocDefault);
	hipMemcpy(way->dev_gfmult_table, gfmult_table, \
		sizeof(gfmult_table), hipMemcpyHostToDevice);

	//��ʼ��������˷����м���
	uint8_t temp[16];
	memset(temp, 0, 16);

	for (int i = 0; i < 16; i++)
	{
		temp[i] ^= add[i];
	}
	multi(gfmult_table, temp);

	uint8_t *gfmult_init = (uint8_t *)malloc(PARTICLE_SIZE);
	for (int i = 0; i < PARTICLE_SIZE / 16; i++)
	{
		memcpy(gfmult_init + i * 16, temp, 16);
	}

	//��ʼ��������˷�����ռ�
	hipHostAlloc((void**)&(way->dev_gfmult), \
		PARTICLE_SIZE, hipHostMallocDefault);

	{
		dim3 grid(GRID_SIZE, 1, 1);
		dim3 block(BLOCK_SIZE, 1, 1);

		for (int i = 0; i < STREAM_SIZE; i++)
		{
			//��������˷���������ȫ���ڴ�
			hipMemcpyAsync(\
				way->dev_gfmult + i * (PARTICLE_SIZE / STREAM_SIZE), \
				gfmult_init + i * (PARTICLE_SIZE / STREAM_SIZE), \
				PARTICLE_SIZE / STREAM_SIZE, \
				hipMemcpyHostToDevice, way->stream[i]);
		}

		for (int i = 0; i < STREAM_SIZE; i++)
		{
			//��ȫ���ڴ��е�GF�˷���������Դ洢ģʽת���ɾ��δ洢ģʽ
			kernal_linear_to_matrix << < grid, block, 0, way->stream[i] >> > (\
				way->dev_matrix_table_zero, way->dev_linear_table_zero, \
				way->dev_matrix_table_one, way->dev_linear_table_one, \
				way->dev_matrix_table_two, way->dev_linear_table_two, \
				way->dev_matrix_table_three, way->dev_linear_table_three, \
				way->dev_gfmult + i * (PARTICLE_SIZE / STREAM_SIZE), \
				way->dev_gfmult + i * (PARTICLE_SIZE / STREAM_SIZE));
		}

		for (int i = 0; i < STREAM_SIZE; i++)
		{
			//ͬ����
			hipStreamSynchronize(way->stream[i]);
		}
	}

	free(gfmult_init);
}

/*
**	�����ӿں���:����������豸�ڴ���ͷŹ�����
*/
void Free_device_memory(device_memory *way)
{
	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//ͬ����
		hipStreamSynchronize(way->stream[i]);
	}

	//�ͷ�ȫ���ڴ�
	hipHostFree(way->dev_gfmult_table);
	hipHostFree(way->dev_IV);
	hipHostFree(way->dev_SboxTable);

	hipHostFree(way->dev_matrix_table_zero);
	hipHostFree(way->dev_linear_table_zero);
	hipHostFree(way->dev_matrix_table_one);
	hipHostFree(way->dev_linear_table_one);
	hipHostFree(way->dev_matrix_table_two);
	hipHostFree(way->dev_linear_table_two);
	hipHostFree(way->dev_matrix_table_three);
	hipHostFree(way->dev_linear_table_three);

	hipHostFree(way->dev_input);
	hipHostFree(way->dev_output);
	hipHostFree(way->dev_gfmult);

	//�ͷ���
	for (int i = 0; i < STREAM_SIZE; i++)
	{
		hipStreamDestroy(way->stream[i]);
	}
}

/*
**	��֤���������ӿں���
**	counter:�������
**	input:��������
**	output:�������
*/
void sm4_gcm_enc(device_memory *way, uint32_t counter, uint8_t input[PARTICLE_SIZE], uint8_t output[PARTICLE_SIZE])
{
	dim3 grid(GRID_SIZE, 1, 1);
	dim3 block(BLOCK_SIZE, 1, 1);

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//�����Ĵ������ڴ濽�����豸ȫ���ڴ�
		hipMemcpyAsync(\
			way->dev_input + i * (PARTICLE_SIZE / STREAM_SIZE), \
			input + i * (PARTICLE_SIZE / STREAM_SIZE), \
			PARTICLE_SIZE / STREAM_SIZE, \
			hipMemcpyHostToDevice, way->stream[i]);
	}

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//���������ݿ���м���
		kernal_enc << < grid, block, 0, way->stream[i] >> > (way->dev_SboxTable, \
			way->dev_matrix_table_zero, way->dev_linear_table_zero, \
			way->dev_matrix_table_one, way->dev_linear_table_one, \
			way->dev_matrix_table_two, way->dev_linear_table_two, \
			way->dev_matrix_table_three, way->dev_linear_table_three, \
			counter, i, \
			way->dev_input + i * (PARTICLE_SIZE / STREAM_SIZE), \
			way->dev_output + i * (PARTICLE_SIZE / STREAM_SIZE));
	}

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//�����ܺ���������ݿ���豸ȫ���ڴ濽���������ڴ�
		hipMemcpyAsync(output + i * (PARTICLE_SIZE / STREAM_SIZE), \
			way->dev_output + i * (PARTICLE_SIZE / STREAM_SIZE), \
			PARTICLE_SIZE / STREAM_SIZE, \
			hipMemcpyDeviceToHost, way->stream[i]);
	}
/*
	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//�����Դ洢ģʽ���������ݿ�ת��Ϊ���δ洢ģʽ
		kernal_linear_to_matrix << < grid, block, 0, way->stream[i] >> > (\
			way->dev_matrix_table_zero, way->dev_linear_table_zero, \
			way->dev_matrix_table_one, way->dev_linear_table_one, \
			way->dev_matrix_table_two, way->dev_linear_table_two, \
			way->dev_matrix_table_three, way->dev_linear_table_three, \
			way->dev_output + i * (PARTICLE_SIZE / STREAM_SIZE), \
			way->dev_input + i * (PARTICLE_SIZE / STREAM_SIZE));
	}

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//����������˷��ͼӷ�����
		kernal_gfmult << < grid, block, 0, way->stream[i] >> > (\
			(uint8_t(*)[256][16])(way->dev_gfmult_table), \
			way->dev_input + i * (PARTICLE_SIZE / STREAM_SIZE), \
			way->dev_gfmult + i * (PARTICLE_SIZE / STREAM_SIZE));
	}
*/
	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//��ͬ��
		hipStreamSynchronize(way->stream[i]);
	}
}

/*
**	��֤���������ӿں���
**	counter:�������
**	input:��������
**	output:�������
*/
void sm4_gcm_dec(device_memory *way, uint32_t counter, uint8_t input[PARTICLE_SIZE], uint8_t output[PARTICLE_SIZE])
{
	dim3 grid(GRID_SIZE, 1, 1);
	dim3 block(BLOCK_SIZE, 1, 1);

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//�����Ĵ������ڴ濽�����豸ȫ���ڴ�
		hipMemcpyAsync(\
			way->dev_input + i * (PARTICLE_SIZE / STREAM_SIZE), \
			input + i * (PARTICLE_SIZE / STREAM_SIZE), \
			PARTICLE_SIZE / STREAM_SIZE, \
			hipMemcpyHostToDevice, way->stream[i]);
	}

	//�����Դ洢ģʽ���������ݿ�ת��Ϊ���δ洢ģʽ
	for (int i = 0; i < STREAM_SIZE; i++)
	{
		kernal_linear_to_matrix << < grid, block, 0, way->stream[i] >> > (\
			way->dev_matrix_table_zero, way->dev_linear_table_zero, \
			way->dev_matrix_table_one, way->dev_linear_table_one, \
			way->dev_matrix_table_two, way->dev_linear_table_two, \
			way->dev_matrix_table_three, way->dev_linear_table_three, \
			way->dev_input + i * (PARTICLE_SIZE / STREAM_SIZE), \
			way->dev_output + i * (PARTICLE_SIZE / STREAM_SIZE));
	}

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//����������˷��ͼӷ�����
		kernal_gfmult << < grid, block, 0, way->stream[i] >> > (\
			(uint8_t(*)[256][16])(way->dev_gfmult_table), \
			way->dev_output + i * (PARTICLE_SIZE / STREAM_SIZE), \
			way->dev_gfmult + i * (PARTICLE_SIZE / STREAM_SIZE));
	}

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//���������ݿ���н���
		kernal_enc << < grid, block, 0, way->stream[i] >> > (way->dev_SboxTable, \
			way->dev_matrix_table_zero, way->dev_linear_table_zero, \
			way->dev_matrix_table_one, way->dev_linear_table_one, \
			way->dev_matrix_table_two, way->dev_linear_table_two, \
			way->dev_matrix_table_three, way->dev_linear_table_three, \
			counter, i, \
			way->dev_input + i * (PARTICLE_SIZE / STREAM_SIZE), \
			way->dev_output + i * (PARTICLE_SIZE / STREAM_SIZE));
	}

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//�����ܺ���������ݿ���豸ȫ���ڴ濽���������ڴ�
		hipMemcpyAsync(output + i * (PARTICLE_SIZE / STREAM_SIZE), \
			way->dev_output + i * (PARTICLE_SIZE / STREAM_SIZE), \
			PARTICLE_SIZE / STREAM_SIZE, \
			hipMemcpyDeviceToHost, way->stream[i]);
	}

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//��ͬ��
		hipStreamSynchronize(way->stream[i]);
	}
}

/*
**	�������ӿں����������յı�ǩ
**	length:�������ݿ鳤��
**	tag:ֵ�������������ִ����Ͻ�������tag���ڴ�ռ�
*/
void sm4_gcm_final(device_memory *way, uint64_t length, uint8_t tag[PARTICLE_SIZE])
{
	uint8_t temp[16];
	/* eor (len(A)||len(C)) */
	uint64_t temp_len = (uint64_t)(16 * 8); // len(A) = (uint64_t)(add_len*8)
	for (int i = 1; i <= 16 / 2; i++)
	{
		temp[16 / 2 - i] = (uint8_t)temp_len;
		temp_len = temp_len >> 8;
	}
	length = length * 16;
	temp_len = (uint64_t)(length * 8); // len(C) = (uint64_t)(length*8)
	for (int i = 1; i <= 16 / 2; i++)
	{
		temp[16 - i] = (uint8_t)temp_len;
		temp_len = temp_len >> 8;
	}

	//��ʼ��(len(A)||len(C))
	hipMemcpyToSymbol(HIP_SYMBOL(constant_lenAC), temp, 16);

	dim3 grid(GRID_SIZE, 1, 1);
	dim3 block(BLOCK_SIZE, 1, 1);

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//�������յ�GHASH���
		kernal_final << < grid, block, 0, way->stream[i] >> > ((uint8_t(*)[256][16])(way->dev_gfmult_table), \
			way->dev_matrix_table_zero, way->dev_linear_table_zero, \
			way->dev_matrix_table_one, way->dev_linear_table_one, \
			way->dev_matrix_table_two, way->dev_linear_table_two, \
			way->dev_matrix_table_three, way->dev_linear_table_three, \
			way->dev_gfmult + i * (PARTICLE_SIZE / STREAM_SIZE));
	}

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//��ȫ���ڴ��о��δ洢ģʽ��GHASH���ת�������Դ洢ģʽ
		kernal_matrix_to_linear << < grid, block, 0, way->stream[i] >> > (\
			way->dev_matrix_table_zero, way->dev_linear_table_zero, \
			way->dev_matrix_table_one, way->dev_linear_table_one, \
			way->dev_matrix_table_two, way->dev_linear_table_two, \
			way->dev_matrix_table_three, way->dev_linear_table_three, \
			way->dev_gfmult + i * (PARTICLE_SIZE / STREAM_SIZE), \
			way->dev_gfmult + i * (PARTICLE_SIZE / STREAM_SIZE));
	}

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//��ÿ���̵߳ı�ǩtag��ȫ���ڴ濽���������ڴ�
		hipMemcpyAsync(tag + i * (PARTICLE_SIZE / STREAM_SIZE), \
			way->dev_gfmult + i * (PARTICLE_SIZE / STREAM_SIZE), \
			PARTICLE_SIZE / STREAM_SIZE, \
			hipMemcpyDeviceToHost, way->stream[i]);
	}

	for (int i = 0; i < STREAM_SIZE; i++)
	{
		//ͬ����
		hipStreamSynchronize(way->stream[i]);
	}
}